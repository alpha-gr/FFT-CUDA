#include "hip/hip_runtime.h"
#include "fft-v2-cuda.cuh"
#include <thrust/complex.h>
#include <complex>
#include <math.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#define rowIndex (num_channels * size * size * blockIdx.y) + (size * size * blockIdx.x) + (size * threadIdx.x)
#define sliceIndex (num_channels * size * size * blockIdx.y) + (size * size * blockIdx.x)

#define FORWARD 1
#define REVERSE -1

#define CHECK(call){\
    const hipError_t error = call;\
    if (error != hipSuccess) {\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("code:%d, reason: %s\n", error,\
            hipGetErrorString(error));\
    }\
}

__device__ void FFT1D(short dir, thrust::complex<float>* data, int length) {

    long n = 1 << length; // 2^length, numero di punti della FFT

    //applica il bit reversal al vettore
    int i2 = n >> 1; // n/2
    int k, j = 0;
    thrust::complex<float> tmp;
    for (int i=0; i<n-1; i++) {
        if (i < j) {
            tmp = data[i];
            data[i] = data[j];
            data[j] = tmp;
        }
        k = i2;
        while (k <= j) {
            j -= k;
            k >>= 1;
        }
        j+=k;
    }

    //FFT (algoritmo cooley-tukey)
    int i, i1, l, l1, l2 = 1;
    thrust::complex<float> u, t;
    thrust::complex<float> c = {-1.0, 0.0};
    for (l=0;l<length;l++) {
        l1 = l2;
        l2 <<= 1;
		u = thrust::complex<float>(1.0, 0.0);
        for (j=0;j<l1;j++) {
            for (i=j;i<n;i+=l2) {
                i1 = i + l1;
                t = u * data[i1];
                data[i1] = data[i] - t;
                data[i] += t;
            }
            u = u * c;
        }

        c.imag(sqrt((1.0 - c.real()) / 2.0));
        if (dir == 1)
            c.imag(-c.imag());
        c.real(sqrt((1.0 + c.real()) / 2.0));
    }

    /* Scaling for forward transform */
    if (dir == 1) {
        for (i=0;i<n;i++) {
            data[i].real(data[i].real()/n);
            data[i].imag(data[i].imag()/n);
        }
    }

}

__global__ void FFT2D_GPU_COMPUTE(thrust::complex<float>* data, int size, int nlog2, int num_slices, int num_channels, short dir) {

    int i, j;
	// FFT delle righe
    FFT1D(dir, data + rowIndex, nlog2);

    __syncthreads();

    thrust::complex<float> tmp;
    // solo il primo thread scambia le righe con le colonne elemento per elemento
    if (threadIdx.x == 0) {
		for (i = 0; i < size; i++) {
			for (j = i; j < size; j++) {
                tmp = data[sliceIndex + i * size + j];
				data[sliceIndex + i * size + j] = data[sliceIndex + j * size + i];
				data[sliceIndex + j * size + i] = tmp;
			}
		}
    }
	__syncthreads();
    //FFT delle colonne
	FFT1D(dir, data + rowIndex, nlog2);

	__syncthreads();

    //shift delle righe e delle colonne
  //  if (threadIdx.x < size / 2) {
		//for (i = 0; i < size / 2; i++) {
		//	tmp = data[sliceIndex + (threadIdx.x * size) + i];
  //          data[sliceIndex + (threadIdx.x * size) + i] = data[sliceIndex + ((threadIdx.x + (size / 2)) * size) + i + (size / 2)];
  //          data[sliceIndex + ((threadIdx.x + (size / 2)) * size) + i + (size / 2)] = tmp;
		//}
  //  }
  //  else {
  //      for (i = 0; i < size/2; i++) {
		//	tmp = data[sliceIndex + (i * size) + threadIdx.x];
		//	data[sliceIndex + (i * size) + threadIdx.x] = data[sliceIndex + ((i + (size / 2)) * size) + threadIdx.x - (size / 2)];
		//	data[sliceIndex + ((i + (size / 2)) * size) + threadIdx.x - (size / 2)] = tmp;
  //      }
  //  }

    for (int i = 0; i < size / 2; i++) {
        //int dstRow = ((threadIdx.x + size / 2) % size);
        //int dstCol = ((i + size / 2) % size);

        tmp = data[sliceIndex + (threadIdx.x * size) + i];
        data[sliceIndex + (threadIdx.x * size) + i] =
            data[sliceIndex + (((threadIdx.x + size / 2) % size) * size) + ((i + size / 2) % size)];
        data[sliceIndex + (((threadIdx.x + size / 2) % size) * size) + ((i + size / 2) % size)] = tmp;
    }

    __syncthreads();
}

__global__ void FFT_SHIFT_GPU(thrust::complex<float>* data, thrust::complex<float>* temp, int n) {

	int n2 = n / 2;
    thrust::complex<float> tmp;
    // Shift delle righe
    for (int i = 0; i < n2; i++) {
		temp[threadIdx.x*n + i + n2] = data[threadIdx.x * n + i];
		temp[threadIdx.x * n + i] = data[threadIdx.x * n + i + n2];
    }

	// Barriera per sincronizzare i thread
    __syncthreads();

    // Shift delle colonne
    for (int i = 0; i < n2; i++) {
		data[i * n + threadIdx.x] = temp[(i + n2) * n + threadIdx.x];
		data[(i + n2) * n + threadIdx.x] = temp[i * n + threadIdx.x];
    }
}

bool FFT2D_GPU(thrust::complex<float>* data, int size, int num_channels, int num_slices, short dir) {

    int nlog2 = log2(size);
    unsigned int data_size = num_slices * num_channels * size * size * sizeof(thrust::complex<float>);

    thrust::complex<float>* data_gpu;
    CHECK(hipMalloc((void**)&data_gpu, data_size));
	
    CHECK(hipMemcpy(data_gpu, data, data_size, hipMemcpyHostToDevice));

    dim3 grid(num_channels, num_slices);
    dim3 block(size);

    FFT2D_GPU_COMPUTE <<<grid, block>>> (data_gpu, size, nlog2, num_slices, num_channels, dir);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(data, data_gpu, data_size, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(data_gpu));

    return true;
}
