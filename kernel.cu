#include "hip/hip_runtime.h"

#pragma once
#include "thrust/complex.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <chrono>
#include "ismrmrd/ismrmrd.h"
#include "ismrmrd/dataset.h"
#include "ismrmrd/serialization.h"
#include "ismrmrd/xml.h"
#include "utils.h"
#include "fft-v2-cuda.cuh"

#define index(slice, ch, row, col, size, n_ch) (n_ch * size * size * slice) + (size * size * ch) + (size * row) + col


using namespace std;

int main() {

    cout << "Lettura del file..." << endl;

    string datafile = "C:/Users/user/source/repos/FFT/mridata/52c2fd53-d233-4444-8bfd-7c454240d314.h5";

    ISMRMRD::Dataset d(datafile.c_str(), "dataset", false);

    unsigned int num_acquisitions = d.getNumberOfAcquisitions();
    cout << "Number of acquisitions: " << num_acquisitions << endl;

    ISMRMRD::Acquisition acq;
    d.readAcquisition(0, acq);
    unsigned int num_channels = acq.active_channels();
    unsigned int num_samples = acq.number_of_samples();
    unsigned int num_slices = num_acquisitions / num_samples;

    // width and height of the slice

    cout << "Number of channels: " << num_channels << endl;
    cout << "Number of samples: " << num_samples << endl;
    cout << "Number of slices: " << num_slices << endl;

    // padded array size to perform FFT
    unsigned int size = next_power_of_two(num_samples);

    cout << "Reading data..." << endl;

    thrust::complex<float>* data;

	data = (thrust::complex<float>*)malloc(size * size * num_slices * num_channels * sizeof(thrust::complex<float>));

	memset(data, 0, size * size * num_slices * num_channels * sizeof(thrust::complex<float>));

    //reading all the data with padding

	complex<float> tmp = complex<float>(0.0, 0.0);
	int pad = (size - num_samples) / 2;

    for (int slice = 0; slice < num_slices; slice++) {
        for (int row = 0; row < num_samples; row++) {
			d.readAcquisition(slice * num_samples + row, acq);
            for (int channel = 0; channel < num_channels; channel++) {
                for (int col = 0; col < num_samples; col++) {
                    tmp = acq.data(col, channel);
					data[index(slice, channel, row+pad, col+pad, size, num_channels)] = thrust::complex<float>(tmp.real(), tmp.imag());
                }
            }
        }
    
    }

	cout << "Processing data..." << endl;

    for (int slice = 0; slice < num_slices; slice++) {

        // 2D IFFT
        auto start = std::chrono::high_resolution_clock::now();
        for (int channel = 0; channel < num_channels; channel++) {

			FFT2D_GPU( data + index(slice, channel, 0, 0, size, num_channels), 512, 1);

            //FFT_SHIFT(slice_channels_padded[channel], padded_width, padded_height);
        }
        auto end = std::chrono::high_resolution_clock::now();
        auto duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Tempo impiegato: " << duration_ms.count() << " millisecondi" << std::endl;


        // final vector to store the image
        vector<vector<float>> mri_image(size, vector<float>(size, 0.0));

        // combine the coils
        for (int row = 0; row < size; ++row) {
            for (int col = 0; col < size; ++col) {
                float sumSquares = 0.0;
                for (int ch = 0; ch < num_channels; ++ch) {
                    // Magnitudine del valore complesso per il coil k
                    float magnitude = abs(data[index(slice,ch,row,col,size,num_channels)]);
                    sumSquares += magnitude * magnitude;
                }
                // Calcola il risultato RSS
                mri_image[row][col] = sqrt(sumSquares);
            }
        }


        // rotate the image by 90 degrees
        rotate_90_degrees(mri_image);

        // flip 
        //flipVertical(mri_image, padded_width, padded_height);
        //flipHorizontal(mri_image, padded_width, padded_height);

        string magnitudeFile = "C:/Users/user/source/repos/FFT-CUDA/output/" + to_string(slice) + ".png";

        write_to_png(mri_image, magnitudeFile);
    } // end for slice


    return 0;

}